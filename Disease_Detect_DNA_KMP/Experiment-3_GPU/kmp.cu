
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string>
#include <time.h>
#include <fstream>
#include <iostream>
using namespace std;


__global__ void KMP(char* pattern, char* text, int prefixTable[], int result[], int pattern_length, int text_length) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int i = pattern_length * index;
    int j = pattern_length * (index + 2) - 1;
    
    if(i > text_length) {
        return;
    }

    if(j > text_length) {
        j = text_length;
    }

    int k = 0;        
    while (i < j) {
        if (k == -1) {
            i++;
            k = 0;
        } else if (text[i] == pattern[k]) {
            i++;
            k++;
            if (k == pattern_length) {
                result[i - pattern_length] = i - pattern_length;
                i = i - k + 1;
            }
        }
        else {
            k = prefixTable[k];
        }
    }
    return;
}

void loadInputFile(string fName, char* inputArray) {
	ifstream inputFile;

	inputFile.open(fName.c_str());
	if (inputFile.is_open()) {
		int cnt = 0;
		while (!inputFile.eof()) {
			string temp;
			getline(inputFile, temp, '\n');
			inputArray[cnt++] = atof(temp.c_str());
		}
		inputFile.close();
	}
}


void preKMP(char* pattern, int prefixTable[]) {
    int m = strlen(pattern);
    int k;
    prefixTable[0] = -1;
    for (int i = 1; i < m; i++) {
        k = prefixTable[i - 1];
        while (k >= 0) {
            if (pattern[k] == pattern[i - 1]) {
                 break;
            }
            else {
                k = prefixTable[k];
            }
        }
        prefixTable[i] = k + 1;
    }
}

int main(int argc, char* argv[]) {
    int textlen = 200000;
	int patternlen = 10;
 
    char* text = (char*)malloc(textlen * sizeof(char));
    char* pattern = (char*)malloc(patternlen * sizeof(char));
  
	std::ifstream file;
	file.open("KMP_Input_200000.txt");
	file.getline(text, textlen);
	file.close();
	file.open("pat.txt");
    file.getline(pattern, 10);
    int text_length = strlen(text);
    int pattern_length = strlen(pattern);

    char *d_text;
    char *d_pattern;

    int *prefixTable,*d_prefixTable;
    int *result,*d_result;
    prefixTable = new int[text_length];
    result = new int[text_length];
   
    for(int i = 0; i < text_length; i++) {
        result[i] = -1;
    }     

    preKMP(pattern, prefixTable);

    hipEvent_t start, stop;
    float elapsedTime;

    hipEventCreate( &start ); 
    hipEventCreate( &stop );

    hipEventRecord( start, 0 );

    hipMalloc((void **)&d_text, text_length * sizeof(char));
    hipMalloc((void **)&d_pattern, pattern_length * sizeof(char));
    hipMalloc((void **)&d_prefixTable, text_length * sizeof(int));
    hipMalloc((void **)&d_result, text_length * sizeof(int));

    hipMemcpy(d_text, text, text_length * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_pattern, pattern, pattern_length * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_prefixTable, prefixTable, text_length * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_result, result, text_length * sizeof(int), hipMemcpyHostToDevice);
    KMP<<<(text_length / pattern_length + 4)/4, 4>>>(d_pattern, d_text ,d_prefixTable, d_result, pattern_length, text_length);

    hipMemcpy(result, d_result, text_length * sizeof(int), hipMemcpyDeviceToHost);

    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    hipEventElapsedTime( &elapsedTime, start, stop );
    hipEventDestroy(start); 
    hipEventDestroy(stop);

    int matches=0;
    for(int i = 0; i < text_length; i++) {
        if (result[i] != -1) {
            matches++;
        }
    }
    cout << "Length of text " << textlen << endl;
	cout << "Length of pattern " << strlen(pattern) << endl;
    cout<<"Number of matches of \""<<pattern<<"\" is "<<matches<<endl<<"Time taken: "<< elapsedTime;
    hipFree(d_text); 
    hipFree(d_pattern); 
    hipFree(d_prefixTable); 
    hipFree(d_result);
    free(text);
    free(pattern);
    return 0;
}